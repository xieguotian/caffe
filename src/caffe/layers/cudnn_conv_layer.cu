
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_conv_layer.hpp"

namespace caffe {
	template <typename Dtype>
	__global__ void mean_vector_clamp(int n_thread, Dtype* input_data,int out_ch, int in_ch, int h, int w)
	{
		CUDA_KERNEL_LOOP(index, n_thread) {
			int spat_idx = index % (w*h);
			int out_ch_idx = index / w / h;

			Dtype sum = 0;
			for (int in_ch_idx = 0; in_ch_idx < in_ch; in_ch_idx++)
			{
				int idx = (out_ch_idx*in_ch + in_ch_idx) *(w*h) + spat_idx;
				sum += input_data[idx];
			}
			sum = sum / in_ch;

			for (int in_ch_idx = 0; in_ch_idx < in_ch; in_ch_idx++)
			{
				int idx = (out_ch_idx*in_ch + in_ch_idx) *(w*h) + spat_idx;
				input_data[idx] -= sum;
				if (input_data[idx]>1)
					input_data[idx] = 1;
				if (input_data[idx] < -1)
					input_data[idx] = -1;
			}
		}
	}

__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
	if (is_incremental_ && (!is_history_init_ ))//|| caffe::is_refresh_incremental))
	{
		int idx_param_idx = this->blobs_.size() - 1;
		caffe_copy(this->blobs_[0]->count(),
			this->blobs_[0]->gpu_data(),
			w_history_.mutable_gpu_data());
		is_history_init_ = true;
		//if (caffe::is_refresh_incremental)
		//{
		//	caffe_gpu_set(this->blobs_[idx_param_idx]->count(), (Dtype)0, this->blobs_[idx_param_idx]->mutable_gpu_data());
		//}
			//this->blobs_[idx_param_idx]->mutable_gpu_data());
	}
	if (is_incremental_)
	{
		int idx_param_idx = this->blobs_.size() - 1;
		caffe_gpu_add(this->blobs_[0]->count(),
			w_history_.gpu_data(),
			this->blobs_[idx_param_idx]->gpu_data(),
			this->blobs_[0]->mutable_gpu_data());
	}

  const Dtype* weight = this->blobs_[0]->gpu_data();

  if (this->layer_param_.convolution_param().is_binarized_param() && !this->blobs_[0]->is_binarized()) 
  {
	  static bool is_first = true;
	  if (is_first)
	  {
		  LOG(INFO) << "use binary weight for training";
		  is_first = false;
	  }
	  int n_thread = this->blobs_[0]->num()*this->blobs_[0]->height()*this->blobs_[0]->width();

	  if (this->phase_ == TRAIN)
	  {
		  // sub mean and clamp to [-1,1]
		  mean_vector_clamp<Dtype> << <CAFFE_GET_BLOCKS(n_thread), CAFFE_CUDA_NUM_THREADS >> > (
			  n_thread, this->blobs_[0]->mutable_gpu_data(), this->blobs_[0]->num(),
			  this->blobs_[0]->channels(), this->blobs_[0]->height(), this->blobs_[0]->width()
			  );
	  }
	  // binarize weight
	  caffe_gpu_sign(this->blobs_[0]->count(), this->blobs_[0]->gpu_data(), sign_weight_.mutable_gpu_data());
	  // calculate abs(weight)
	  caffe_gpu_abs<Dtype>(this->blobs_[0]->count(), this->blobs_[0]->gpu_data(), sign_weight_.mutable_gpu_diff());
	  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
		  this->blobs_[0]->num(), 1,
		  this->blobs_[0]->channels()*this->blobs_[0]->height()*this->blobs_[0]->width(),
		  (Dtype)1.0 / this->blobs_[0]->num(),
		  //this->blobs_[0]->gpu_data(),
		  sign_weight_.gpu_diff(),
		  sum_cache_.gpu_data(),
		  (Dtype)0.0,
		  sum_result_.mutable_gpu_data()
		  );
	  
	  //approximate real weight.
	  num_mul_kernel<Dtype> << <CAFFE_GET_BLOCKS(this->blobs_[0]->count()), CAFFE_CUDA_NUM_THREADS >> >(
		  this->blobs_[0]->count(), this->blobs_[0]->channels(), this->blobs_[0]->height()* this->blobs_[0]->width(),
		  sign_weight_.gpu_data(), sum_result_.gpu_data(), sign_weight_.mutable_gpu_data()
		  );
	  weight = sign_weight_.gpu_data();
  }

  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }

  if (is_direct_connect_)
  {
	  int idx_param_idx = this->blobs_.size() - 1;
	  for (int i = 0; i < bottom.size(); ++i)
	  {
		  const Dtype* bottom_data = bottom[i]->gpu_data();
		  Dtype* top_data = top[i]->mutable_gpu_data();
		  for (int idx = 0; idx < this->blobs_[idx_param_idx]->count(); ++idx)
		  {
			  int sel_idx = this->blobs_[idx_param_idx]->cpu_data()[idx];
			  for (int n = 0; n < bottom[i]->num(); ++n)
			  {
				  caffe_copy(bottom[i]->count(2),
					  bottom_data + bottom[i]->offset(n, sel_idx),
					  top_data + top[i]->offset(n, idx + num_output_));
			  }
		  }
	  }
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	const Dtype* weight = NULL;
	Dtype* weight_diff = NULL;
	if (this->param_propagate_down_[0]) {
		weight = this->blobs_[0]->gpu_data();
		weight_diff = this->blobs_[0]->mutable_gpu_diff();
	}
	Dtype* bias_diff = NULL;
	if (this->bias_term_ && this->param_propagate_down_[1]) {
		bias_diff = this->blobs_[1]->mutable_gpu_diff();
	}
	for (int i = 0; i < top.size(); ++i) {
		const Dtype* top_diff = top[i]->gpu_diff();
		const Dtype* top_data = top[i]->gpu_data();
		// Backward through cuDNN in parallel over groups and gradients.
		for (int g = 0; g < this->group_; g++) {
			// Gradient w.r.t. bias.
			if (this->bias_term_ && this->param_propagate_down_[1]) {
				CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0 * this->group_ + g],
					cudnn::dataType<Dtype>::one,
					top_descs_[i], top_diff + top_offset_ * g,
					cudnn::dataType<Dtype>::one,
					bias_desc_, bias_diff + bias_offset_ * g));
			}

			// Gradient w.r.t. weights.
			if (this->param_propagate_down_[0]) {
				const Dtype* bottom_data = bottom[i]->gpu_data();
				CUDNN_CHECK(cudnnConvolutionBackwardFilter(
					handle_[1 * this->group_ + g],
					cudnn::dataType<Dtype>::one,
					bottom_descs_[i], bottom_data + bottom_offset_ * g,
					top_descs_[i], top_diff + top_offset_ * g,
					conv_descs_[i],
					bwd_filter_algo_[i], workspace[1 * this->group_ + g],
					workspace_bwd_filter_sizes_[i],
					cudnn::dataType<Dtype>::one,
					filter_desc_, weight_diff + this->weight_offset_ * g));
			}

			// Gradient w.r.t. bottom data.
			if (propagate_down[i]) {
				if (weight == NULL) {
					weight = this->blobs_[0]->gpu_data();
				}
				if (this->layer_param_.convolution_param().is_binarized_param())
				{
					static bool is_first = true;
					if (is_first)
					{
						LOG(INFO) << "use binary weight for backward";
						is_first = false;
					}
					weight = sign_weight_.gpu_data();
				}
				Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
				CUDNN_CHECK(cudnnConvolutionBackwardData(
					handle_[2 * this->group_ + g],
					cudnn::dataType<Dtype>::one,
					filter_desc_, weight + this->weight_offset_ * g,
					top_descs_[i], top_diff + top_offset_ * g,
					conv_descs_[i],
					bwd_data_algo_[i], workspace[2 * this->group_ + g],
					workspace_bwd_data_sizes_[i],
					cudnn::dataType<Dtype>::zero,
					bottom_descs_[i], bottom_diff + bottom_offset_ * g));
			}
		}

		// Synchronize the work across groups, each of which went into its own
		// stream, by launching an empty kernel into the default (null) stream.
		// NOLINT_NEXT_LINE(whitespace/operators)
		sync_conv_groups << <1, 1 >> >();
	}

	Dtype decay_mult = this->layer_param_.decay_mult();
	if (decay_mult > 0)
	{
		for (int i = 0; i < top.size(); ++i) {
			const Dtype* top_diff = top[i]->gpu_diff();
			const Dtype* top_data = top[i]->gpu_data();
			// Backward through cuDNN in parallel over groups and gradients.
			for (int g = 0; g < this->group_; g++) {
				// Gradient w.r.t. weights.
				if (this->param_propagate_down_[0]) {
					const Dtype* bottom_data = bottom[i]->gpu_data();
					// backpropagate  signal decay.
					CUDNN_CHECK(cudnnConvolutionBackwardFilter(
						handle_[1 * this->group_ + g],
						//cudnn::dataType<Dtype>::one,
						&decay_mult,
						bottom_descs_[i], bottom_data + bottom_offset_ * g,
						//top_descs_[i], top_diff + top_offset_ * g,
						top_descs_[i], top_data + top_offset_*g,
						conv_descs_[i],
						bwd_filter_algo_[i], workspace[1 * this->group_ + g],
						workspace_bwd_filter_sizes_[i],
						cudnn::dataType<Dtype>::one,
						filter_desc_, weight_diff + this->weight_offset_ * g));
				}
			}
			sync_conv_groups << <1, 1 >> >();
		}
	}

  if (is_direct_connect_)
  {
	  int idx_param_idx = this->blobs_.size() - 1;
	  for (int i = 0; i < bottom.size(); ++i)
	  {
		  Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
		  const Dtype* top_diff = top[i]->gpu_diff();
		  for (int idx = 0; idx < this->blobs_[idx_param_idx]->count(); ++idx)
		  {
			  int sel_idx = this->blobs_[idx_param_idx]->cpu_data()[idx];
			  for (int n = 0; n < bottom[i]->num(); ++n)
			  {
				  caffe_gpu_axpy(bottom[i]->count(2), (Dtype)1.0,
					  top_diff + top[i]->offset(n, idx + num_output_),
					  bottom_diff + bottom[i]->offset(n, sel_idx));
			  }
		  }
	  }
  }
  if (is_incremental_)
  {
	  int idx_param_idx = this->blobs_.size() - 1;
	  caffe_copy(this->blobs_[0]->count(),
		  this->blobs_[0]->gpu_diff(),
		  this->blobs_[idx_param_idx]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
