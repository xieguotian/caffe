#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/normalize_layer.hpp"

namespace caffe
{
	template <typename Dtype>
	__global__ void NormalizeForward(const int n, const Dtype* in, Dtype* out, 
		Dtype* norm_cache_data, int height, int width, int channels,int num,Dtype eps)
	{
		CUDA_KERNEL_LOOP(index, n) {
			int w = index % width;
			int h = (index / width) % height;
			int nn = (index / width / height) % num;

			for (int ch = 0; ch < channels; ++ch)
			{
				int idx = ((nn*channels + ch)*height + h)*width + w;
				norm_cache_data[index] += in[idx];
			}
			norm_cache_data[index] += eps;
			for (int ch = 0; ch < channels; ++ch)
			{
				int idx = ((nn*channels + ch)*height + h)*width + w;
				out[idx] = in[idx] / norm_cache_data[index];
			}
		}
	}

	template <typename Dtype>
	void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		/*
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* norm_cache_data = norm_cache_.mutable_gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();
		
		int count = norm_cache_.count();
		caffe_gpu_set(count, (Dtype)0, norm_cache_data);
		const Dtype eps = (Dtype)std::numeric_limits<Dtype>::epsilon();
		NormalizeForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, bottom_data, top_data,norm_cache_data, bottom[0]->height(), bottom[0]->width(),
			bottom[0]->channels(), bottom[0]->num(),eps);


		CUDA_POST_KERNEL_CHECK;
		*/

		Dtype* norm_cache_data = norm_cache_.mutable_gpu_data();
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();

		caffe_gpu_set(norm_cache_.count(), (Dtype)0, norm_cache_data);
		int tmp_offset = norm_cache_.offset(1);
		for (int n = 0; n < bottom[0]->num(); ++n)
		{
			for (int ch = 0; ch < bottom[0]->channels(); ++ch)
			{
				caffe_gpu_axpy(tmp_offset, (Dtype)1,
					bottom_data, norm_cache_data);
				bottom_data += tmp_offset;
			}
			norm_cache_data += tmp_offset;
		}
		caffe_gpu_add_scalar(norm_cache_.count(), (Dtype)std::numeric_limits<Dtype>::epsilon(), norm_cache_data);

		norm_cache_data = norm_cache_.mutable_gpu_data();
		bottom_data = bottom[0]->gpu_data();

		for (int n = 0; n < bottom[0]->num(); ++n)
		{
			for (int ch = 0; ch < bottom[0]->channels(); ++ch)
			{
				caffe_gpu_div(tmp_offset, bottom_data, norm_cache_data, top_data);
				bottom_data += tmp_offset;
				top_data += tmp_offset;
			}
			norm_cache_data += tmp_offset;
		}
	}

	template <typename Dtype>
	__global__ void NormalizeBackward(const int n, const Dtype* in_data, Dtype* out_diff,const Dtype* in_diff,
		const Dtype* norm_cache_data, int height, int width, int channels, int num)
	{
		CUDA_KERNEL_LOOP(index, n) {
			int w = index % width;
			int h = (index / width) % height;
			int nn = (index / width / height) % num;
			Dtype val = 0;
			for (int ch = 0; ch < channels; ++ch)
			{
				int idx = ((nn*channels + ch)*height + h)*width + w;
				val += in_diff[idx] * in_data[idx] / norm_cache_data[index] / norm_cache_data[index];
			}
			for (int ch = 0; ch < channels; ++ch)
			{
				int idx = ((nn*channels + ch)*height + h)*width + w;
				out_diff[idx] = in_diff[idx] / norm_cache_data[index] - val;
			}
		}
	}
	template <typename Dtype>
	void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
	{
		/*
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* norm_cache_data = norm_cache_.gpu_data();
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

		//int count = bottom[0]->count();
		int count = norm_cache_.count();
		NormalizeBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, bottom_data, bottom_diff, top_diff, norm_cache_data, bottom[0]->height(),
			bottom[0]->width(), bottom[0]->channels(), bottom[0]->num());
		*/
		Dtype* norm_cache_diff = norm_cache_.mutable_gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		const Dtype* top_diff = top[0]->gpu_diff();
		const Dtype* norm_cache_data = norm_cache_.gpu_data();
		const Dtype* bottom_data = bottom[0]->gpu_data();

		caffe_gpu_mul(bottom[0]->count(), top_diff, bottom_data, bottom_diff);
		caffe_gpu_set(norm_cache_.count(), (Dtype)0, norm_cache_diff);
		int tmp_offset = norm_cache_.offset(1);
		for (int n = 0; n < bottom[0]->num(); ++n)
		{
			for (int ch = 0; ch < bottom[0]->channels(); ++ch)
			{
				caffe_gpu_div(tmp_offset, bottom_diff, norm_cache_data, bottom_diff);
				caffe_gpu_div(tmp_offset, bottom_diff, norm_cache_data, bottom_diff);
				caffe_gpu_axpy(tmp_offset, (Dtype)1, bottom_diff, norm_cache_diff);
				bottom_diff += tmp_offset;
			}
			norm_cache_data += tmp_offset;
			norm_cache_diff += tmp_offset;
		}

		norm_cache_diff = norm_cache_.mutable_gpu_diff();
		norm_cache_data = norm_cache_.gpu_data();
		bottom_diff = bottom[0]->mutable_gpu_diff();
		for (int n = 0; n < bottom[0]->num(); ++n)
		{
			for (int ch = 0; ch < bottom[0]->channels(); ++ch)
			{
				caffe_gpu_div(tmp_offset, top_diff, norm_cache_data, bottom_diff);
				caffe_gpu_sub(tmp_offset, bottom_diff, norm_cache_diff, bottom_diff);
				bottom_diff += tmp_offset;
				top_diff += tmp_offset;
			}
			norm_cache_data += tmp_offset;
			norm_cache_diff += tmp_offset;
		}

	}

	INSTANTIATE_LAYER_GPU_FUNCS(NormalizeLayer);
}