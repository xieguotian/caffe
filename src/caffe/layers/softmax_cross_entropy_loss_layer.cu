#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_cross_entropy_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void cross_entropy_kernel(const int n, const Dtype* a, 
	const Dtype* b, Dtype* y, Dtype* counts,
	const bool has_ignore_label_, const int ignore_label_, 
	const int spatial_dim, const int channels,
	const Dtype* label) {
	CUDA_KERNEL_LOOP(index, n) {
		const int n = index / spatial_dim / channels;
		const int s = index % spatial_dim;
		int label_value;
		if (label!=NULL)
			label_value = static_cast<int>(label[n * spatial_dim + s]);

		if (label!=NULL && has_ignore_label_ && label_value != ignore_label_) {
			y[index] = 0;
			counts[index] = 0;
		}
		else{
			y[index] = -b[index] * log(max(a[index], Dtype(FLT_MIN)));
			counts[index] = 1;
		}
	}
}

template <typename Dtype>
void SoftmaxCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();

  Dtype* counts = prob_.mutable_gpu_diff();
  const Dtype* hard_label = NULL;
  if (bottom.size() == 3)
	  hard_label = bottom[2]->gpu_data();

  cross_entropy_kernel<Dtype> << <bottom[0]->count(), CAFFE_CUDA_NUM_THREADS >> >(
	  bottom[0]->count(),
	  prob_data, label, 
	  loss_data, counts, 
	  has_ignore_label_, ignore_label_,
	  inner_num_, bottom[0]->channels(),
	  hard_label);

  Dtype loss;
  caffe_gpu_asum(bottom[0]->count(), loss_data, &loss);

  Dtype valid_count = outer_num_*inner_num_;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
	  has_ignore_label_ && bottom.size() == 3) {
	  caffe_gpu_asum(prob_.count(), counts, &valid_count);
	  valid_count /= bottom[0]->channels();
  }

  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_,
	  valid_count);
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}


template <typename Dtype>
void SoftmaxCrossEntropyLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* label = bottom[1]->gpu_data();

	caffe_gpu_sub(bottom[0]->count(), prob_data, label, bottom_diff);
	if (has_ignore_label_ && bottom.size() == 3)
		caffe_gpu_mul(bottom[0]->count(), bottom_diff, prob_.gpu_diff(), bottom_diff);
	if (use_T_)
	{
		caffe_gpu_scal(bottom[0]->count(), (Dtype)1.0 / temperature_, bottom_diff);
	}

	// Since this memory is never used for anything else,
	// we use to to avoid allocating new GPU memory.
	Dtype* counts = prob_.mutable_gpu_diff();

	Dtype valid_count = outer_num_*inner_num_;
	// Only launch another CUDA kernel if we actually need the count of valid
	// outputs.
	if (normalization_ == LossParameter_NormalizationMode_VALID &&
		has_ignore_label_ && bottom.size() == 3) {
		caffe_gpu_asum(prob_.count(), counts, &valid_count);
		valid_count /= bottom[0]->channels();
	}

    const Dtype loss_weight = top[0]->cpu_diff()[0] /
		get_normalizer(normalization_, valid_count);
    caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxCrossEntropyLossLayer);

}  // namespace caffe
