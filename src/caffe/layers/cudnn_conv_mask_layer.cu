
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_conv_mask_layer.hpp"

namespace caffe {

__global__ void sync_conv_groups_t() { }

template <typename Dtype>
__global__ void max_among_six_spatial(const int nthreads,
	const Dtype* const input_data, 
	const int num, const int channels,
	const int height, const int width,
	Dtype* const output_data,char* const output_mask)
{
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int w_idx = index % width;
		const int h_idx = (index / width) % height;
		const int ch_idx = (index / height / width) % channels;
		const int num_idx = index / channels / height / width;

		Dtype d[9];

		Dtype max_value = -FLT_MAX;
		int max_pos = -1;
		int g_idx;
		int tmp_w_idx;
		int tmp_h_idx;
		for (int i = 0; i < 3; i++)
		{
			for (int j = 0; j < 3; j++)
			{
				g_idx = i * 3 + j;
				tmp_w_idx = j - 1 + w_idx;
				tmp_h_idx = i - 1 + h_idx;
				if (tmp_w_idx < 0 || tmp_w_idx >= width || tmp_h_idx < 0 || tmp_h_idx >= height)
					d[g_idx] = 0;
				else
					d[g_idx] = input_data[(((num_idx*9+g_idx)*channels+ch_idx)*height+tmp_h_idx)*width+tmp_w_idx];
			}
		}
		
		//Dtype val[6];

		//val[0] = d[4];
		//val[1] = d[3] + d[4] + d[5];
		//val[2] = d[1] + d[4] + d[7];
		//val[3] = d[0] + d[4] + d[8];
		//val[4] = d[2] + d[4] + d[6];
		//val[5] = d[0] + d[1] + d[2] + d[3] + d[4] + d[5] + d[6] + d[7] + d[8];
		//max_value = d[0] + d[1] + d[2] + d[3];

		d[0] = d[0] + d[4] + d[8];
		d[1] = d[1] + d[4] + d[7];
		d[2] = d[2] + d[4] + d[6];
		d[3] = d[3] + d[4] + d[5];
		//d[5] = max_value + d[4] + d[5] + d[6] + d[7] + d[8];
		d[5] = d[0] + d[1] + d[2] + d[3] - 3 * d[4];

		//max_value = val[0];
		max_value = d[0];
		max_pos = 0;

		for (int i = 1; i < 6; i++)
		{
			//if (max_value < val[i])
			if (max_value < d[i])
			{
				//max_value = val[i];
				max_value = d[i];
				max_pos = i;
			}
		}
		output_data[index] = max_value;
		output_mask[index] = (char)max_pos;

	}
}

template <typename Dtype>
void CuDNNConvolutionMaskLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();

  shared_ptr<Blob<Dtype>> caches_;
  caches_ = thread_caches_[thread_id_];
  
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();

	Dtype* top_data = caches_->mutable_gpu_data(); //top[i]->mutable_gpu_data(); 
	// Forward through cuDNN in parallel over groups.
	for (int g = 0; g < this->group_; g++) {
		// Filters.
		CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
			cudnn::dataType<Dtype>::one,
			bottom_descs_[i], bottom_data + bottom_offset_ * g,
			filter_desc_, weight + this->weight_offset_ * g,
			conv_descs_[i],
			fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
			cudnn::dataType<Dtype>::zero,
			top_descs_[i], top_data + top_offset_ * g));

		// Bias.
		if (this->bias_term_) {
			const Dtype* bias_data = this->blobs_[1]->gpu_data();
			CUDNN_CHECK(cudnnAddTensor(handle_[g],
				cudnn::dataType<Dtype>::one,
				bias_desc_, bias_data + bias_offset_ * g,
				cudnn::dataType<Dtype>::one,
				top_descs_[i], top_data + top_offset_ * g));
		}
	}

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups_t<<<1, 1>>>();

	// forward data from cacehs to top
	int n_threads = top[i]->count();
	//Dtype* mask_data = top[i * 2 + 1]->mutable_gpu_data();
	char* mask_data = mask_caches_[i]->mutable_gpu_data();
	max_among_six_spatial<Dtype> << <CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS >> >(
		n_threads, top_data, top[i]->num(), top[i]->channels(), top[i]->height(), top[i]->width(),
		top[i]->mutable_gpu_data(), mask_data
		);
  }

}

template <typename Dtype>
__global__ void max_among_six_spatial_bp(const int nthreads,
	const Dtype* const input_diff,
	const int num, const int channels,
	const int height, const int width,
	Dtype* const output_diff, const char* const output_mask)
{

	CUDA_KERNEL_LOOP(index, nthreads) {
		const int w_idx = index % width + 1;
		const int h_idx = (index / width) % height + 1;
		const int ch_idx = (index / height / width) % channels;
		const int g_idx = (index / height / width / channels) % 9;
		const int num_idx = index / channels / height / width / 9;

		//int j = g_idx % 3;
		//int i = g_idx / 3;

		int w = w_idx - (int)(g_idx % 3);//j;
		int h = h_idx - (int)(g_idx / 3);//i;

		if (w >= 0 && w < width && h >= 0 && h < height)
		{
			int idx = ((num_idx*channels + ch_idx)*height + h)*width + w;
			int sel_num = output_mask[idx];
			switch (sel_num)
			{
			//case 0:
			case 4:
				if (g_idx == 4)
					output_diff[index] = input_diff[idx];
				break;
			//case 1:
			case 3:
				if (g_idx == 3 || g_idx == 4 || g_idx == 5)
					output_diff[index] = input_diff[idx];
				break;
			//case 2:
			case 1:
				if (g_idx == 1 || g_idx == 4 || g_idx == 7)
					output_diff[index] = input_diff[idx];
				break;
			//case 3:
			case 0:
				if (g_idx == 0 || g_idx == 4 || g_idx == 8)
					output_diff[index] = input_diff[idx];
				break;
			//case 4:
			case 2:
				if (g_idx == 2 || g_idx == 4 || g_idx == 6)
					output_diff[index] = input_diff[idx];
				break;
			case 5:
				output_diff[index] = input_diff[idx];
				break;
			}
		}

		//Dtype val = 0;
		//for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
		//	for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
		//		int idx = ((num_idx*channels + ch_idx)*height + h_col)*width + w_col;
		//		int sel_num = output_mask[idx];
		//		switch (sel_num)
		//		{
		//		case 0:
		//			if (g_idx == 4)
		//				val += input_diff[idx];
		//			break;
		//		case 1:
		//			if (g_idx == 3 || g_idx == 4 || g_idx == 5)
		//				val += input_diff[idx];
		//			break;
		//		case 2:
		//			if (g_idx == 1 || g_idx == 4 || g_idx == 7)
		//				val += input_diff[idx];
		//			break;
		//		case 3:
		//			if (g_idx == 0 || g_idx == 4 || g_idx == 8)
		//				val += input_diff[idx];
		//			break;
		//		case 4:
		//			if (g_idx == 2 || g_idx == 4 || g_idx == 6)
		//				val += input_diff[idx];
		//			break;
		//		case 5:
		//			val += input_diff[idx];
		//			break;
		//		}
		//	}
		//}
		//output_diff[index] = val;

		//Dtype val = 0;
		//const int w_im = index % width + 1;
		//const int h_im = (index / width) % height + 1;
		//const int c_im = index / (width * height);
		//const int n_idx = index / (width*height*channels * 9);
		//
		//int kernel_extent_w = 3;//kernel_w;
		//int kernel_extent_h = 3;// kernel_h;
		//// compute the start and end of the output
		//const int w_col_start =
		//	(w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) + 1;
		//const int w_col_end = min(w_im  + 1, width_col);
		//const int h_col_start =
		//	(h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) + 1;
		//const int h_col_end = min(h_im  + 1, height_col);

		//// TODO: use LCM of stride and dilation to avoid unnecessary loops
		//for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
		//	for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
		//		int h_k = (h_im - h_col);
		//		int w_k = (w_im - w_col);
		//		//int data_col_index = (((c_im * 3 + h_k) * 3 + w_k) *
		//		//	height_col + h_col) * width_col + w_col;
		//		int data_col_index = 
		//		val += input_diff[data_col_index];
		//	}
		//}
		//output_diff[index] = val;

		//const int w_idx = index % width;
		//const int h_idx = (index / width) % height;
		//const int ch_idx = (index / height / width) % channels;
		//const int num_idx = index / channels / height / width;
		//int sel_idx[9];
		//int num_sel = 0;
		//switch (output_mask[index])
		//{
		//case 0:
		//	sel_idx[0] = 4;
		//	num_sel = 1;
		//	break;
		//case 1:
		//	//int sel_idx[3] = { 3, 4, 5 };
		//	sel_idx[0] = 3; sel_idx[1] = 4; sel_idx[2] = 5;
		//	num_sel = 3;
		//	break;
		//case 2:
		//	//int sel_idx[3] = { 1, 4, 7 };
		//	sel_idx[0] = 1; sel_idx[1] = 4; sel_idx[2] = 7;
		//	num_sel = 3;
		//	break;
		//case 3:
		//	//int sel_idx[3] = { 0, 4, 8 };
		//	sel_idx[0] = 0; sel_idx[1] = 4; sel_idx[2] = 8;
		//	num_sel = 3;
		//	break;
		//case 4:
		//	//int sel_idx[3] = { 2, 4, 6 };
		//	sel_idx[0] = 2; sel_idx[1] = 4; sel_idx[2] = 6;
		//	num_sel = 3;
		//	break;
		//case 5:
		//	//int sel_idx[9] = { 0, 1, 2, 3, 4, 5, 6, 7, 8 };
		//	for (int i = 0; i < 9; i++)
		//		sel_idx[i] = i;
		//	num_sel = 9;
		//	break;
		//}
		//for (int i = 0; i < 9; i++)
		//{
		//	int g_idx = i;
		//	int tmp_w_idx = int(g_idx % 3) - 1 + w_idx;
		//	int tmp_h_idx = int(g_idx / 3) - 1 + h_idx;
		//	if (tmp_w_idx < 0 || tmp_w_idx >= width || tmp_h_idx < 0 || tmp_h_idx >= height)
		//		continue;
		//	output_diff[(((num_idx * 9 + g_idx)*channels + ch_idx)*height + tmp_h_idx)*width + tmp_w_idx] = 0.1*input_diff[index];
		//}
		//for (int i = 0; i < num_sel; i++)
		//{
		//	int g_idx = sel_idx[i];
		//	int tmp_w_idx = int(g_idx % 3) - 1 + w_idx;
		//	int tmp_h_idx = int(g_idx / 3) - 1 + h_idx;
		//	if (tmp_w_idx < 0 || tmp_w_idx >= width || tmp_h_idx < 0 || tmp_h_idx >= height)
		//		continue;
		//	output_diff[(((num_idx * 9 + g_idx)*channels + ch_idx)*height + tmp_h_idx)*width + tmp_w_idx] = input_diff[index];
		//}
	}
}

template <typename Dtype>
void CuDNNConvolutionMaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  shared_ptr<Blob<Dtype>> caches_;
  caches_ = thread_caches_[thread_id_];

  for (int i = 0; i < top.size(); ++i) {

	  // propagate diff to caches.
	  int n_threads = top[i]->count() * 9;
	  caffe_gpu_set(n_threads, (Dtype)0, caches_->mutable_gpu_data());
	  const char* mask_data = mask_caches_[i]->gpu_data();
	  max_among_six_spatial_bp<Dtype> << <CAFFE_GET_BLOCKS(n_threads), CAFFE_CUDA_NUM_THREADS >> >(
		  n_threads, top[i]->gpu_diff(), top[i]->num(),
		  top[i]->channels(), top[i]->height(), top[i]->width(),
		  caches_->mutable_gpu_data(), mask_data); 
	  const Dtype* top_diff = caches_->gpu_data();//top[i]->gpu_diff();

    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups_t<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionMaskLayer);

}  // namespace caffe
#endif
