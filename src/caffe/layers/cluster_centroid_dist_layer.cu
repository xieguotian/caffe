#include "hip/hip_runtime.h"
#include "caffe/layers/cluster_centroid_dist_layer.hpp"
#include <vector>

namespace caffe{
	//template <typename Dtype>
	//__global__ void eucliean_vec_forward(const int n, const int num, const int num_cluster,
	//	const int feat_dim, const Dtype* bottom_data, const Dtype* centroid_data,
	//	Dtype* top_data)
	//{
	//	CUDA_KERNEL_LOOP(index, n) {
	//		int k_idx = n % num_cluster;
	//		int n_idx = n / num_cluster;

	//		const Dtype* bottom_ptr = bottom_data + n_idx*feat_dim;
	//		const Dtype* centroid_ptr = centroid_data + k_idx*feat_dim;
	//		Dtype* top_ptr = top_data + n;
	//		top_ptr[0] = 0;
	//		for (int i = 0; i < feat_dim; ++i)
	//		{
	//			top_ptr[0] += 0.5*(bottom_ptr[i] - centroid_ptr[i])*(bottom_ptr[i] - centroid_ptr[i]);
	//		}

	//	}
	//}
	template <typename Dtype>
	__global__ void set_diag_zero(const int n, Dtype* y) {
		CUDA_KERNEL_LOOP(index, n) {
			int idx = index*n + index;
			y[idx] = 0;
		}
	}

	template <typename Dtype>
	__global__ void delete_diag(const int n, const int sqrtN, Dtype* x, Dtype* y) {
		CUDA_KERNEL_LOOP(index, n) {
			//int idx = index*n + index;
			//y[idx] = 0;
			int dimx = index % sqrtN;
			int dimy = index / sqrtN;
			if (dimx != dimy)
			{
				if (dimx > dimy)
				{
					int idx = dimy*(sqrtN - 1) + dimx - 1;
					y[idx] = x[index];
				}
				else{
					int idx = dimy*(sqrtN - 1) + dimx;
					y[idx] = x[index];
				}
			}
			else
			{
				x[index] = 0;
			}
		}
	}
	template <typename Dtype>
	__global__ void expand_diag(const int n, const int sqrtN, Dtype* x, const Dtype* y) {
		CUDA_KERNEL_LOOP(index, n) {
			//int idx = index*n + index;
			//y[idx] = 0;
			int dimx = index % sqrtN;
			int dimy = index / sqrtN;
			if (dimx != dimy)
			{
				if (dimx > dimy)
				{
					int idx = dimy*(sqrtN - 1) + dimx - 1;
					x[index] = y[idx];
				}
				else{
					int idx = dimy*(sqrtN - 1) + dimx;
					x[index] = y[idx];
				}
			}
			else{
				x[index] = 0;
			}
		}
	}
	template <typename Dtype>
	void ClusterCentroidDistLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{ 
		//caffe_gpu_mul(bottom[0]->count(), bottom[0]->gpu_data(), bottom[0]->gpu_data(), bottom_cache_.mutable_gpu_data());
		//caffe_gpu_gemm(
		//	CblasNoTrans,
		//	CblasNoTrans,
		//	bottom[0]->num(),
		//	1,
		//	centroid_dim_,
		//	(Dtype)1.0,
		//	bottom_cache_.gpu_data(),
		//	ones_.gpu_data(),
		//	(Dtype)0.0,
		//	bottom_cache_.mutable_gpu_diff()
		//	);
		//caffe_gpu_powx(bottom[0]->num(), bottom_cache_.gpu_diff(), (Dtype)0.5, bottom_cache_.mutable_gpu_diff());
		//caffe_gpu_gemm(
		//	CblasNoTrans, CblasNoTrans,
		//	bottom[0]->num(),
		//	centroid_dim_,
		//	1,
		//	(Dtype)1.0,
		//	bottom_cache_.gpu_diff(),
		//	ones_.gpu_data(),
		//	(Dtype)0.0,
		//	bottom_cache_.mutable_gpu_data()
		//	);
		//caffe_gpu_div(bottom[0]->count(), bottom[0]->gpu_data(), bottom_cache_.gpu_data(), bottom_cache_.mutable_gpu_diff());
		//if (use_T_)
		//	caffe_gpu_scal(bottom[0]->count(), (Dtype)T, bottom_cache_.mutable_gpu_diff());

		//const Dtype* bottom_data = bottom_cache_.gpu_diff(); //= bottom[0]->gpu_data();
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = is_self_dist_ ? top_cache_.mutable_gpu_data() : top[0]->mutable_gpu_data();
		const Dtype* centroid_data = compute_dist_ ? bottom[1]->gpu_data() : this->blobs_[0]->gpu_data();
		const int count_blobs_ = compute_dist_ ? bottom[1]->count() : this->blobs_[0]->count();
		const int top_count = is_self_dist_ ? top_cache_.count() : top[0]->count();

		if (!compute_dist_)
		{
			if (is_sample_base_cls)
			{
				caffe_copy(bottom[0]->count(), bottom_data, this->blobs_[0]->mutable_gpu_data());
				for (int i = 0; i < top[1]->count(); ++i)
				{
					top[1]->mutable_cpu_data()[i] = i;
				}
			}
			else
			{
				if (!initialized_)
				{
					if (init_count_ >= this->blobs_[0]->count())
					{
						initialized_ = true;
						LOG(INFO) << "intial centroid complete.";
					}
					else
					{
						int count = min(bottom[0]->count(), this->blobs_[0]->count() - init_count_);
						caffe_copy(count, bottom_data, this->blobs_[0]->mutable_gpu_data() + init_count_);
						caffe_rng_gaussian<Dtype>(top[0]->count(), Dtype(0),
							Dtype(1), top[0]->mutable_cpu_data());
						init_count_ += count;
						LOG(INFO) << init_count_;
						return;
					}
				}
			}
		}
		// square of data.
		caffe_gpu_mul(bottom[0]->count(),
			bottom_data,
			bottom_data,
			square_feat_.mutable_gpu_data());
		// sum along centroid_dim_
		caffe_gpu_gemm(
			CblasNoTrans,
			CblasNoTrans,
			num_samp_,
			//bottom[0]->num(),
			1,
			centroid_dim_,
			(Dtype)0.5, // / centroid_dim_,
			square_feat_.gpu_data(),
			ones_.gpu_data(),
			(Dtype)0.0,
			column_.mutable_gpu_data());
		// span along num_cluster_ dim
		caffe_gpu_gemm(
			CblasNoTrans,
			CblasNoTrans,
			//bottom[0]->num(),
			num_samp_,
			num_cluster_,
			1,
			(Dtype)1.0,
			column_.gpu_data(),
			ones_.gpu_data(),
			(Dtype)0.0,
			top_data
			);
		// dot product of centroid and feat
		caffe_gpu_gemm(
			CblasNoTrans,
			CblasTrans,
			//bottom[0]->num(),
			num_samp_,
			num_cluster_,
			centroid_dim_,
			(Dtype)-1.0, // / centroid_dim_,
			bottom_data,
			centroid_data,
			(Dtype)1.0,
			top_data);


		//square of centroid.
		caffe_gpu_mul(
			count_blobs_, 
			centroid_data, 
			centroid_data, 
			square_cluster_.mutable_gpu_data());

		//sum along centroid_dim_
		caffe_gpu_gemm(
			CblasNoTrans,
			CblasNoTrans, 
			num_cluster_, 
			1,
			centroid_dim_,
			(Dtype)1.0, // / centroid_dim_,
			square_cluster_.gpu_data(),
			ones_.gpu_data(),
			(Dtype)0.0,
			column_.mutable_gpu_data());
		//span along feat num
		caffe_gpu_gemm(CblasNoTrans,
			CblasNoTrans, 
			//bottom[0]->num(),
			num_samp_,
			num_cluster_, 
			1,
			(Dtype)0.5,
			ones_.gpu_data(), 
			column_.gpu_data(),
			(Dtype)1.0,
			top_data);//cache_cluster_.mutable_gpu_data());

		//caffe_gpu_powx(top[0]->count(), top_data, (Dtype)0.5, top_data);
		//caffe_gpu_scal(top[0]->count(), (Dtype)scale, top_data);
		if (!use_square_)
			caffe_gpu_powx(top_count, top_data, (Dtype)0.5, top_data);
		caffe_gpu_scal(top_count, (Dtype)scale, top_data);
		//if (compute_dist_ && bottom[0]->data() == bottom[1]->data())
		if (is_self_dist_)
		{
			/*set_diag_zero<Dtype> << <CAFFE_GET_BLOCKS(top[0]->num()), CAFFE_CUDA_NUM_THREADS >> >(top[0]->num(), top_data);*/
			//delete_diag<Dtype> << <CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS >> >(top_count, top[0]->num(), top_data, top[0]->mutable_gpu_data());
			set_diag_zero<Dtype> << <CAFFE_GET_BLOCKS(top[0]->num()), CAFFE_CUDA_NUM_THREADS >> >(top[0]->num(), top_data);
			caffe_copy(top[0]->count(), top_data, top[0]->mutable_gpu_data());
			
		}
	}


	template <typename Dtype>
	void ClusterCentroidDistLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
	{ 
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff(); 
		Dtype* centroid_diff = compute_dist_ ? bottom[1]->mutable_gpu_diff() : this->blobs_[0]->mutable_gpu_diff();

		if (!compute_dist_)
		{
			if (is_sample_base_cls)
			{
			}
			else{
				if (!initialized_)
				{
					caffe_gpu_set(bottom[0]->count(), (Dtype)0, bottom_diff);
					caffe_gpu_set(this->blobs_[0]->count(), (Dtype)0, centroid_diff);
					return;
				}
			}
		}

		//const Dtype* top_diff = temp_diff_.gpu_data();
		//caffe_gpu_div(top[0]->count(), top[0]->gpu_diff(), top[0]->gpu_data(), top_cache_.mutable_gpu_data());
		//caffe_gpu_scal(top[0]->count(), (Dtype)(scale / 2.0), top_cache_.mutable_gpu_data());

		//if (compute_dist_ && bottom[0]->data() == bottom[1]->data())
		//{
		//	set_diag_zero<Dtype> << <CAFFE_GET_BLOCKS(top_cache_.num()), CAFFE_CUDA_NUM_THREADS >> >(top_cache_.num(), top_cache_.mutable_gpu_data());
		//}
		const int top_count = is_self_dist_ ? top_cache_.count() : top[0]->count();
		if (is_self_dist_)
		{
			//expand_diag<Dtype> << <CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS >> >(top_count, top[0]->num(), top_cache_.mutable_gpu_diff(), top[0]->gpu_diff());
			caffe_copy(top[0]->count(), top[0]->gpu_diff(), top_cache_.mutable_gpu_diff());
			if (!use_square_)
			{
				caffe_gpu_div(top_count, top_cache_.gpu_diff(), top_cache_.gpu_data(), top_cache_.mutable_gpu_diff());
				caffe_gpu_scal(top_count, (Dtype)(scale / 2.0), top_cache_.mutable_gpu_diff());
			}
			set_diag_zero<Dtype> << <CAFFE_GET_BLOCKS(top_cache_.num()), CAFFE_CUDA_NUM_THREADS >> >(top_cache_.num(), top_cache_.mutable_gpu_diff());
		}
		else{
			if (!use_square_)
			{
				caffe_gpu_div(top[0]->count(), top[0]->gpu_diff(), top[0]->gpu_data(), top_cache_.mutable_gpu_data());
				caffe_gpu_scal(top[0]->count(), (Dtype)(scale / 2.0), top_cache_.mutable_gpu_data());
			}
			else
				caffe_copy(top[0]->count(), top[0]->gpu_diff(), top_cache_.mutable_gpu_data());
		}
		const Dtype* top_diff = is_self_dist_ ? top_cache_.gpu_diff() : top_cache_.gpu_data(); //top[0]->gpu_diff();
		//const Dtype* top_diff = top[0]->gpu_diff();
		const Dtype* centroid_data = compute_dist_ ? bottom[1]->gpu_data() : this->blobs_[0]->gpu_data();
		const Dtype* top_data = is_self_dist_ ? top_cache_.gpu_data() : top[0]->gpu_data();
		
		const int count_blobs_ = compute_dist_ ? bottom[1]->count() : this->blobs_[0]->count();
		//const Dtype* bottom_data = bottom_cache_.gpu_diff(); //= bottom[0]->gpu_data();
		const Dtype* bottom_data = bottom[0]->gpu_data();

		//**************propagate diff to centroid***************
		// dot top_diff with feat data.
		if ((compute_dist_ && propagate_down[1]) || (!compute_dist_ && this->param_propagate_down_[0])){
			caffe_gpu_gemm(
				CblasTrans,
				CblasNoTrans,
				num_cluster_,
				centroid_dim_,
				top[0]->num(),
				(Dtype)scale,// / centroid_dim_,
				top_diff,
				//bottom[0]->gpu_data(),
				bottom_data,
				(Dtype)0.0,
				square_cluster_.mutable_gpu_data()
				);

			// sum top_diff along num
			caffe_gpu_gemm(
				CblasNoTrans,
				CblasNoTrans,
				1,
				num_cluster_,
				top[0]->num(),
				(Dtype)scale, // / centroid_dim_,
				ones_.gpu_data(),
				top_diff,
				(Dtype)0.0,
				column_.mutable_gpu_data()
				);
			// expand top_diff along centroid_dim_.
			caffe_gpu_gemm(
				CblasNoTrans,
				CblasNoTrans,
				num_cluster_,
				centroid_dim_,
				1,
				(Dtype)1.0,
				column_.gpu_data(),
				ones_.gpu_data(),
				(Dtype)0.0,
				//centroid_diff
				square_cluster_.mutable_gpu_diff()
				);

			// multipy with centroid data
			//caffe_gpu_mul(this->blobs_[0]->count(), centroid_diff, centroid_data, centroid_diff);
			caffe_gpu_mul(count_blobs_, square_cluster_.mutable_gpu_diff(), centroid_data, square_cluster_.mutable_gpu_diff());
			// sum all diff
			caffe_gpu_sub(count_blobs_, square_cluster_.mutable_gpu_diff(), square_cluster_.gpu_data(), square_cluster_.mutable_gpu_diff());
			caffe_gpu_add(count_blobs_, square_cluster_.mutable_gpu_diff(), centroid_diff, centroid_diff);
		}

		if (propagate_down[0])
		{
			//**************propagate diff to feat data**************
			// dot top_diff with centroid data
			caffe_gpu_gemm(
				CblasNoTrans,
				//CblasTrans,
				CblasNoTrans,
				top[0]->num(),
				centroid_dim_,
				num_cluster_,
				(Dtype)scale, // / centroid_dim_,
				top_diff,
				centroid_data,
				(Dtype)0.0,
				square_feat_.mutable_gpu_data()
				);
			// sum top_diff along num_cluster_
			caffe_gpu_gemm(
				CblasNoTrans,
				CblasNoTrans,
				top[0]->num(),
				1,
				num_cluster_,
				(Dtype)scale,// / centroid_dim_,
				top_diff,
				ones_.gpu_data(),
				(Dtype)0.0,
				column_.mutable_gpu_data()
				);
			//expand top_diff along centroid_dim_.
			caffe_gpu_gemm(
				CblasNoTrans,
				CblasNoTrans,
				top[0]->num(),
				centroid_dim_,
				1,
				(Dtype)1.0,
				column_.gpu_data(),
				ones_.gpu_data(),
				(Dtype)0.0,
				bottom_diff
				);
			//multipy with centroid data.
			caffe_gpu_mul(bottom[0]->count(), bottom_diff, bottom_data/*bottom[0]->gpu_data()*/, bottom_diff);
			//sum all diff
			caffe_gpu_sub(bottom[0]->count(), bottom_diff, square_feat_.gpu_data(), bottom_diff);
		}

		//Dtype scalar_factor = (top[0]->asum_diff() / top[0]->count()) / (bottom[0]->asum_diff() / bottom[0]->count());//top_cache_.asum_data();
		//LOG(INFO) << "scale:" << scalar_factor << "," << bottom[0]->cpu_diff()[0] << "," << top[0]->cpu_diff()[0] << "," << top[0]->cpu_data()[0];
		//caffe_gpu_scal(top[0]->count(), (Dtype)scalar_factor, bottom[0]->mutable_gpu_diff());

		//caffe_gpu_div(bottom[0]->count(), bottom_diff, bottom_cache_.gpu_data(), bottom_diff);
		//caffe_gpu_mul(bottom[0]->count(), bottom_diff, bottom_cache_.gpu_diff(), bottom_cache_.mutable_gpu_data());
		//caffe_gpu_gemm(
		//	CblasNoTrans,
		//	CblasNoTrans,
		//	bottom[0]->num(),
		//	1,
		//	centroid_dim_,
		//	(Dtype)1.0,
		//	bottom_cache_.gpu_data(),
		//	ones_.gpu_data(),
		//	(Dtype)0.0,
		//	column_.mutable_gpu_data()
		//	);
		//caffe_gpu_gemm(
		//	CblasNoTrans,
		//	CblasNoTrans,
		//	bottom[0]->num(),
		//	centroid_dim_,
		//	1,
		//	(Dtype)1.0,
		//	column_.gpu_data(),
		//	ones_.gpu_data(),
		//	(Dtype)0.0, 
		//	bottom_cache_.mutable_gpu_data()
		//	);
		//caffe_gpu_mul(bottom[0]->count(), bottom_cache_.gpu_diff(), bottom_cache_.gpu_data(), bottom_cache_.mutable_gpu_data());
		//if (use_T_)
		//{
		//	caffe_gpu_scal(bottom[0]->count(), (Dtype)(1.0 / T), bottom_cache_.mutable_gpu_data());
		//	caffe_gpu_scal(bottom[0]->count(), (Dtype)( T), bottom_diff);
		//}
		//caffe_gpu_sub(bottom[0]->count(), bottom_diff, bottom_cache_.gpu_data(), bottom_diff);

		////////debug
		//for (int i = 0; i < 20; ++i)
		//	LOG(INFO) <<"#"<<i<<": " << this->blobs_[0]->cpu_data()[i] << "," << this->blobs_[0]->cpu_diff()[i] << ","
		//	<< bottom_cache_.cpu_diff()[i] <<"," <<bottom[0]->cpu_data()[i] << "," << bottom[0]->cpu_diff()[i] << "," << top[0]->cpu_data()[i] << "," << top[0]->cpu_diff()[i];

		//caffe_gpu_gemm(
		//	CblasNoTrans, 
		//	CblasNoTrans,
		//	top[0]->num(), 
		//	centroid_dim_, 
		//	num_cluster_,
		//	(Dtype)1.0*scale, 
		//	top_diff, 
		//	this->blobs_[1]->gpu_data(), 
		//	(Dtype)0.0, 
		//	square_feat_.mutable_gpu_data());
		////multipy feat data
		//caffe_gpu_mul(bottom[0]->count(), 
		//	square_feat_.gpu_data(), 
		//	bottom[0]->gpu_data(), 
		//	square_feat_.mutable_gpu_data());


		//caffe_gpu_mul(
		//	this->blobs_[0]->count(), 
		//	centroid_data, 
		//	this->blobs_[1]->gpu_data(), 
		//	square_cluster_.mutable_gpu_data());

		////dot diff of feat
		//caffe_gpu_gemm(
		//	CblasNoTrans, 
		//	CblasNoTrans, 
		//	top[0]->num(), 
		//	centroid_dim_, num_cluster_,
		//	(Dtype)-1.0*scale, 
		//	top_diff, 
		//	square_cluster_.gpu_data(), 
		//	(Dtype)0.0, bottom_diff);

		////dot diff of centroid //problem?
		//caffe_gpu_gemm(CblasTrans,
		//	CblasNoTrans, 
		//	num_cluster_, 
		//	centroid_dim_, 
		//	top[0]->num(),
		//	(Dtype)-1.0*scale, 
		//	top_diff, top_data, 
		//	(Dtype)0.0, 
		//	centroid_diff);

		//caffe_gpu_mul(this->blobs_[0]->count(),
		//	centroid_diff, 
		//	this->blobs_[1]->gpu_data(), 
		//	centroid_diff);


		////sum diff along feat num
		//caffe_gpu_gemm(CblasNoTrans, 
		//	CblasNoTrans, 
		//	1, 
		//	num_cluster_,
		//	top[0]->num(),
		//	(Dtype)1.0*scale, 
		//	ones_.gpu_data(), 
		//	top_diff,(Dtype)0.0, 
		//	column_.mutable_gpu_data());
		////span diff along centroid dim
		//caffe_gpu_gemm(CblasNoTrans, 
		//	CblasNoTrans,
		//	num_cluster_, 
		//	centroid_dim_, 
		//	1,
		//	(Dtype)1.0, 
		//	column_.gpu_data(), 
		//	ones_.gpu_data(),
		//	(Dtype)0.0, 
		//	square_cluster_.mutable_gpu_data());

		////multipy centroid data
		//caffe_gpu_mul(
		//	this->blobs_[0]->count(),
		//	square_cluster_.gpu_data(),
		//	this->blobs_[0]->gpu_data(), 
		//	square_cluster_.mutable_gpu_data());
		////multiply std normalizar.
		//caffe_gpu_mul(
		//	this->blobs_[0]->count(), 
		//	square_cluster_.gpu_data(), 
		//	this->blobs_[1]->gpu_data(), 
		//	square_cluster_.mutable_gpu_data());


		////sum all diff.
		//caffe_gpu_add(
		//	bottom[0]->count(),
		//	bottom_diff, 
		//	square_feat_.gpu_data(), 
		//	bottom_diff);
		//caffe_gpu_add(
		//	this->blobs_[0]->count(), 
		//	centroid_diff,
		//	square_cluster_.gpu_data(), 
		//	centroid_diff);

	}
	INSTANTIATE_LAYER_GPU_FUNCS(ClusterCentroidDistLayer);
}